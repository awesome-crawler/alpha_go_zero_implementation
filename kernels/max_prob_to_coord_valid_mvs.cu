#include "hip/hip_runtime.h"
__global__ void max_prob_to_coord_valid_mvs_kernel(float * prob_map, int * to_coord, 
		char * board, char * valid_mv_map_internal){
	int gm = blockIdx.x;
	int gm_offset = gm*MAP_SZ;
	float * prob_map_cur = &prob_map[gm_offset];

	COUNT_VALID

	// determine max prob
	float max_prob = -999;
	int max_map_loc;
	for(int mv_ind = 0; mv_ind < n_valid_mvs; mv_ind++){ // skip pass move
		int map_loc = valid_mv_inds[mv_ind];
		CHK_VALID_MV_MAP_COORD(map_loc)
		DASSERT(map_loc == MAP_SZ || board[gm*MAP_SZ + map_loc] == 0)
		if(prob_map_cur[map_loc] <= max_prob)
			continue;
		max_map_loc = map_loc;
		max_prob = prob_map_cur[map_loc];
	}

	to_coord[gm] = max_map_loc;
}

void max_prob_to_coord_valid_mvs_launcher(float * prob_map, int * to_coord){
	hipError_t err;
	REQ_INIT

	max_prob_to_coord_valid_mvs_kernel <<< BATCH_SZ, 1 >>> (prob_map, to_coord, board, 
		valid_mv_map_internal); CHECK_CUDA_ERR

	VERIFY_BUFFER_INTEGRITY
}


