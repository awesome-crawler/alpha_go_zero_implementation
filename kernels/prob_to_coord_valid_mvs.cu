#include "hip/hip_runtime.h"
__global__ void prob_to_coord_valid_mvs_kernel(float * prob_map, int * to_coord, 
		char * board, hiprandState_t* rand_states, char * valid_mv_map_internal){
	int gm = blockIdx.x;
	float * prob_map_cur = &prob_map[gm*(MAP_SZ+1)];

	int gm_offset = gm*MAP_SZ;
	COUNT_VALID
	
	float rand_val = (float)(hiprand(&rand_states[gm]) % RAND_RES);
	rand_val /= (float)RAND_RES;

	// compute probs sum over valid mvs
	float probs_sum_orig = 0;
	for(int mv_ind = 0; mv_ind < n_valid_mvs; mv_ind++){
		int map_loc = valid_mv_inds[mv_ind];
		CHK_VALID_MV_MAP_COORD(map_loc)
		DASSERT(map_loc == MAP_SZ || board[gm*MAP_SZ + map_loc] == 0)
		probs_sum_orig += prob_map_cur[map_loc];
	}
	if(probs_sum_orig == 0) probs_sum_orig = 1;
	//assert(probs_sum_orig >= 0);
	
	float probs_sum = 0;
	for(int mv_ind = 0; mv_ind < n_valid_mvs; mv_ind++){
		int map_loc = valid_mv_inds[mv_ind];
		float p = prob_map_cur[map_loc] / probs_sum_orig;
		//if(!(p >= 0 && p <= 1))
		//	printf("prob err %f\n", p);
		//DASSERT(p >= 0 && p <= 1)

		// randomly selected or we're at the last move
		if(((rand_val >= probs_sum) && (rand_val < (probs_sum + p)))){// || 
		//		(mv_ind == (n_valid_mvs - 1))){
			to_coord[gm] = map_loc;
			return;
		}
		probs_sum += p;
	}

	assert(0);
}

void prob_to_coord_valid_mvs_launcher(float * prob_map, int * to_coord){
	hipError_t err;
	REQ_INIT

	prob_to_coord_valid_mvs_kernel <<< BATCH_SZ, 1 >>> (prob_map, to_coord, board, rand_states, valid_mv_map_internal); CHECK_CUDA_ERR

	VERIFY_BUFFER_INTEGRITY
}


